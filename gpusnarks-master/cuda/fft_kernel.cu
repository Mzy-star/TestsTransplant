#include "hip/hip_runtime.h"
/*****************************************************************************
 Implementation of Fast Fourier Transformation on Finite Elements
 *****************************************************************************
 * @author     Marius van der Wijden
 * Copyright [2019] [Marius van der Wijden]
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "fft_kernel.h"
#include "device_field.h"
#include "device_field_operators.h"

#define LOG_NUM_THREADS 10
#define NUM_THREADS (1 << LOG_NUM_THREADS)
#define LOG_CONSTRAINTS 16
#define CONSTRAINTS (1 << LOG_CONSTRAINTS)

#define CUDA_CALL( call )               \
{                                       \
hipError_t result = call;              \
if ( hipSuccess != result )            \
    std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

__device__ __forceinline__
size_t bitreverse(size_t n, const size_t l)
{
    return __brevll(n) >> (64ull - l); 
}

__device__ uint32_t _mod [SIZE] = { 610172929, 1586521054, 752685471, 3818738770, 
    2596546032, 1669861489, 1987204260, 1750781161, 3411246648, 3087994277, 
    4061660573, 2971133814, 2707093405, 2580620505, 3902860685, 134068517, 
    1821890675, 1589111033, 1536143341, 3086587728, 4007841197, 270700578, 764593169, 115910};

template<typename FieldT>  
__global__ void cuda_fft(FieldT *out, FieldT *field) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t log_m = LOG_CONSTRAINTS;
    const size_t length = CONSTRAINTS;
    const size_t block_length = 1ul << (log_m - LOG_NUM_THREADS) ;
    const size_t startidx = idx * block_length;
    assert (CONSTRAINTS == 1ul<<log_m);
    if(startidx > length)
        return;
    FieldT a [block_length];

    //TODO algorithm is non-deterministic because of padding
    FieldT omega_j = FieldT(_mod);
    omega_j = omega_j ^ idx; // pow
    FieldT omega_step = FieldT(_mod);
    omega_step = omega_step ^ (idx << (log_m - LOG_NUM_THREADS));
    
    FieldT elt = FieldT::one();
    //Do not remove log2f(n), otherwise register overflow
    size_t n = block_length, logn = log2f(n);
    assert (n == (1u << logn));
    for (size_t i = 0; i < 1ul<<(log_m - LOG_NUM_THREADS); ++i)
    {
        const size_t ri = bitreverse(i, logn);
        for (size_t s = 0; s < NUM_THREADS; ++s)
        {
            // invariant: elt is omega^(j*idx)
            size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) % (1u << log_m);
            FieldT tmp = field[id];
            tmp = tmp * elt;
            if (s != 0) tmp = tmp + a[ri];
            a[ri] = tmp;
            elt = elt * omega_step;
        }
        elt = elt * omega_j;
    }

    const FieldT omega_num_cpus = FieldT(_mod) ^ NUM_THREADS;
    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        const FieldT w_m = omega_num_cpus^(n/(2*m));
        for (size_t k = 0; k < n; k += 2*m)
        {
            FieldT w = FieldT::one();
            for (size_t j = 0; j < m; ++j)
            {
                const FieldT t = w;
                w = w * a[k+j+m];
                a[k+j+m] = a[k+j] - t;
                a[k+j] = a[k+j] + t;
                w = w * w_m;
            }
        }
        m = m << 1;
    }
    for (size_t j = 0; j < 1ul<<(log_m - LOG_NUM_THREADS); ++j)
    {
        if(((j << LOG_NUM_THREADS) + idx) < length)
            out[(j<<LOG_NUM_THREADS) + idx] = a[j];
    }
}

template<typename FieldT> 
void best_fft (std::vector<FieldT> &a, const FieldT &omg)
{
	int cnt;
    hipGetDeviceCount(&cnt);
    printf("CUDA Devices: %d, Field size: %lu, Field count: %lu\n", cnt, sizeof(FieldT), a.size());
    assert(a.size() == CONSTRAINTS);

    size_t blocks = NUM_THREADS / 256 + 1;
    size_t threads = NUM_THREADS > 256 ? 256 : NUM_THREADS;
    printf("NUM_THREADS %u, blocks %lu, threads %lu \n",NUM_THREADS, blocks, threads);

    FieldT *in;
    CUDA_CALL( hipMalloc((void**)&in, sizeof(FieldT) * a.size()); )
    CUDA_CALL( hipMemcpy(in, (void**)&a[0], sizeof(FieldT) * a.size(), hipMemcpyHostToDevice); )

    FieldT *out;
    CUDA_CALL( hipMalloc(&out, sizeof(FieldT) * a.size()); )
    cuda_fft<FieldT> <<<blocks,threads>>>(out, in);
        
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    CUDA_CALL( hipMemcpy((void**)&a[0], out, sizeof(FieldT) * a.size(), hipMemcpyDeviceToHost); )

    CUDA_CALL( hipDeviceSynchronize();)
}

//List with all templates that should be generated
template void best_fft(std::vector<fields::Scalar> &v, const fields::Scalar &omg);
